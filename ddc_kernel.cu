#include <complex>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cstdio> // 使用 C++ 风格的头文件

using namespace std;
static constexpr float PI=3.14159265358979323846;

// DDC 处理所需的 GPU 资源
struct DDCResources
{
    int N;  // 每次追加的数据长度
    int M; // 累积多少块数据后计算
    int NDEC;
    int K;
    int16_t *d_indata;
    hipFloatComplex *d_outdata;
    hipFloatComplex *gpu_buffer;
    float *d_fir_coeffs;
    int16_t *h_indata;
    int h_index;
};

// 复数乘法
__device__ static hipFloatComplex complex_mult(float a, float b, float c, float d)
{
    return make_hipFloatComplex(a * c - b * d, a * d + b * c);
}

__global__ void mix(int16_t *indata, hipFloatComplex *gpu_buffer, int offset, int N, int M, int lo_ch)
{
    int total_size=N*M;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_size)
    {
        float phase=-(float)(i%N)*(float)lo_ch/(float)N*2.0*PI;
        float lo_cos=cos(phase);
        float lo_sin=sin(phase);
        gpu_buffer[offset + i] = complex_mult(float(indata[i]), 0.0f, lo_cos, lo_sin);
    }
}

// 设备核函数：FIR 滤波并下抽样
__global__ void fir_filter(hipFloatComplex *gpu_buffer, hipFloatComplex *outdata, const float *fir_coeffs, int NDEC, int K, int total_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int output_index = i;
    int input_index = i * NDEC;

    if (output_index < total_size / NDEC)
    {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int j = 0; j < K * NDEC; j++)
        {
            sum = hipCaddf(sum, hipCmulf(make_hipFloatComplex(fir_coeffs[j], 0.0f), gpu_buffer[input_index + j]));
        }
        outdata[output_index] = sum;
    }
}

// 初始化 DDC 资源
void init_ddc_resources(DDCResources *res,int N, int M, int NDEC, int K, const float *fir_coeffs)
{
    res->NDEC = NDEC;
    res->K = K;
    res->N=N;
    res->M=M;
    int buffer_size = M * N + NDEC * (K - 1);
    int fir_size = NDEC * K;

    hipError_t err = hipMalloc((void **)&res->d_indata, M * N * sizeof(int16_t));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->d_outdata, (M * N / NDEC) * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->gpu_buffer, buffer_size * sizeof(hipFloatComplex));
    assert(err == hipSuccess);
    err = hipMalloc((void **)&res->d_fir_coeffs, fir_size * sizeof(float));
    assert(err == hipSuccess);

    res->h_indata = (int16_t *)malloc(M * N * sizeof(int16_t));
    assert(res->h_indata);
    res->h_index = 0;

    err = hipMemcpy(res->d_fir_coeffs, fir_coeffs, fir_size * sizeof(float), hipMemcpyHostToDevice);
    assert(err == hipSuccess);
}

// 释放资源
void free_ddc_resources(DDCResources *res)
{
    hipFree(res->d_indata);
    hipFree(res->d_outdata);
    hipFree(res->gpu_buffer);
    hipFree(res->d_fir_coeffs);
    free(res->h_indata);
}

// DDC 处理
int ddc(const int16_t *indata, int lo_ch, DDCResources *res)
{
    memcpy(res->h_indata + res->h_index, indata, res->N * sizeof(int16_t));
    res->h_index += res->N;

    if (res->h_index == res->M * res->N)
    {
        int total_size = res->M * res->N;
        //int buffer_size = total_size + res->NDEC * (res->K - 1);
        int offset = res->NDEC * (res->K - 1);

        hipMemcpy(res->d_indata, res->h_indata, total_size * sizeof(int16_t), hipMemcpyHostToDevice);
        mix<<<(total_size + 255) / 256, 256>>>(res->d_indata, res->gpu_buffer, offset, res->N, res->M, lo_ch);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            return -1;
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;

        fir_filter<<<(total_size / res->NDEC + 255) / 256, 256>>>(res->gpu_buffer, res->d_outdata, res->d_fir_coeffs, res->NDEC, res->K, total_size);
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;
        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess)
            return -1;

        
        res->h_index = 0;
        return 1;
    }
    return 0;
}

void fetch_output(std::complex<float> *outdata, DDCResources* res){
    int total_size = res->M * res->N;
    hipMemcpy(outdata, res->d_outdata, (total_size / res->NDEC) * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
}


int calc_output_size(const DDCResources* res){
    return (res->M)*(res->N)/(res->NDEC);
}
